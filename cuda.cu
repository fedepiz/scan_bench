#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <vector>
#include <common.h>


// Helper for checking CUDA error codes
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void cuda_block_scans(float *g_odata, float *g_sums, float *g_idata) {
    __shared__ float temp[BLOCK_SIZE];

    // Downsweep
    int thid = threadIdx.x;
    int block_offset = 2 * blockIdx.x * blockDim.x;
    int offset = 1;

    // Load input in shared memory
    temp[2 * thid] = g_idata[block_offset + 2 * thid];
    temp[2 * thid + 1] = g_idata[block_offset + 2 * thid + 1];

    // Upsweep
    for (int d = BLOCK_SIZE >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (thid < d) {
             int ai = offset*(2*thid+1)-1;     
             int bi = offset*(2*thid+2)-1;
             temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    // clear the last element & record block sum
     if (thid == 0) {
         g_sums[blockIdx.x] = temp[BLOCK_SIZE-1];
         temp[BLOCK_SIZE - 1] = 0; 
    }

    // Downsweep
     for (int d = 1; d < BLOCK_SIZE; d *= 2) {
         offset >>= 1;
         __syncthreads();
         if (thid < d) {
             int ai = offset * (2 * thid + 1) - 1;
             int bi = offset * (2 * thid + 2) - 1;
             float t = temp[ai];
             temp[ai] = temp[bi];
             temp[bi] += t;
         }
     }
     __syncthreads();

     // Write out output
     g_odata[block_offset + 2 * thid] = temp[2 * thid];
     g_odata[block_offset + 2 * thid + 1] = temp[2 * thid + 1];
}

// Feed back the block sums into each partial.
// WARNING: This is a trivial implementation. For reasonable perf, do coalescing!
__global__ void cuda_add(float* partials, float* block_sums, int block_size) {
    auto value = block_sums[blockIdx.x];
    auto chunk = partials + (block_size * blockIdx.x);

    for (auto i = 0; i < block_size; i++) {
        chunk[i] += value;
    }
}

TestRun CudaAlgo::run(std::vector<float> input, bool gold_silent) const {
    auto input_size = input.size();

    const int NUM_BLOCKS = input_size / BLOCK_SIZE;

    const int REMAINDER = input_size % BLOCK_SIZE;
    if (REMAINDER != 0) {
        std::cout << "WARNING: Input size not divisible by block size" << std::endl;
    }

    // Compute the Gold value by running a local scan
    std::vector<float> gold = input;
    local_scan_inplace(gold);


    // Allocate device memory
    float* d_input;
    float *d_output;
    float *d_block_sums;
    gpuErrchk(hipMalloc((void **)&d_input, sizeof(float) * input_size));
    gpuErrchk(hipMalloc((void **)&d_output, sizeof(float) * input_size));
    gpuErrchk(hipMalloc((void **)&d_block_sums, sizeof(float) * NUM_BLOCKS));

    // Load input to GPU
    gpuErrchk(hipMemcpy(d_input, input.data(), sizeof(float) * input_size, hipMemcpyHostToDevice));
    
    // Execute local block scans
    hipEvent_t cuda_block_scans_start, cuda_block_scans_end;
    gpuErrchk(hipEventCreate(&cuda_block_scans_start));
    gpuErrchk(hipEventCreate(&cuda_block_scans_end));

    gpuErrchk(hipEventRecord(cuda_block_scans_start));
    this->block_scans(input_size, d_output, d_block_sums, d_input);
    //cuda_block_scans<<<input_size/BLOCK_SIZE, BLOCK_SIZE/2>>>(d_output, d_block_sums, d_input);
    gpuErrchk(hipEventRecord(cuda_block_scans_end));


    gpuErrchk(hipDeviceSynchronize());

    std::vector<float> test(input_size);
    gpuErrchk(hipMemcpy(test.data(), d_output, sizeof(float) * input_size, hipMemcpyDeviceToHost));
    gpuErrchk(hipDeviceSynchronize());


    // Load block sums to host, sequentially scan, re-upload
    std::vector<float> block_sums(NUM_BLOCKS);
    gpuErrchk(hipMemcpy(block_sums.data(), d_block_sums, sizeof(float) * NUM_BLOCKS, hipMemcpyDeviceToHost));
    local_scan_inplace(block_sums);
    gpuErrchk(hipMemcpy(d_block_sums, block_sums.data(), sizeof(float) * NUM_BLOCKS, hipMemcpyHostToDevice));

    // Add in the block sums
    gpuErrchk(hipDeviceSynchronize());
    hipEvent_t cuda_add_start, cuda_add_end;
    gpuErrchk(hipEventCreate(&cuda_add_start));
    gpuErrchk(hipEventCreate(&cuda_add_end));

    gpuErrchk(hipEventRecord(cuda_add_start));
    cuda_add<<<NUM_BLOCKS, 1>>>(d_output, d_block_sums, BLOCK_SIZE);
    gpuErrchk(hipEventRecord(cuda_add_end));
    gpuErrchk(hipDeviceSynchronize());

    // Copy final output to device
    gpuErrchk(hipMemcpy(input.data(), d_output, sizeof(float) * input_size, hipMemcpyDeviceToHost));

    // Cleanup after kernel execution
    gpuErrchk(hipFree(d_input));
    gpuErrchk(hipFree(d_output));
    gpuErrchk(hipFree(d_block_sums));

    // Collect timing 
    float block_scans_time;
    gpuErrchk(hipEventElapsedTime(&block_scans_time, cuda_block_scans_start, cuda_block_scans_end));
    float add_time;
    gpuErrchk(hipEventElapsedTime(&add_time, cuda_add_start, cuda_add_end));
    Timing timing(block_scans_time, add_time);

    // Perform gold checking
    GoldCheck gold_check(input, gold);
    gold_check.silent = gold_silent;

    // Done!
    return TestRun(std::move(input), timing, gold_check);
}

const char* NvidiaAlgo::name() const {
    return "cuda nvidia";
}

void NvidiaAlgo::block_scans(size_t input_size, float* d_output, float* d_block_sums, float* d_input) const  {
    cuda_block_scans<<<input_size/BLOCK_SIZE, BLOCK_SIZE/2>>>(d_output, d_block_sums, d_input);
}


const char* DpiaAlgo::name() const {
    return "cuda dpia";
}

__global__ void dpia_block_scans_internal(float *output, float *d_block_sums, const float *d_input);

void DpiaAlgo::block_scans(size_t input_size, float *d_output, float *d_block_sums, float *d_input) const {
        dpia_block_scans_internal<<<input_size/BLOCK_SIZE, BLOCK_SIZE/2>>>(d_output, d_block_sums, d_input);
}


__global__
void dpia_block_scans_internal(float* output, float* d_block_sums, const float* d_input){
  /* Start of moved local vars */
  /* End of moved local vars */
  /* mapWorkGroup */
  __shared__ float up1[32];
  __shared__ float up2[16];
  __shared__ float up3[8];
  __shared__ float up4[4];
  __shared__ float up5[2];
  __shared__ float up6[1];

  __shared__ float down1[1];
  __shared__ float down2[2];
  __shared__ float down3[4];
  __shared__ float down4[8];
  __shared__ float down5[16];
  __shared__ float down6[32];

  //for (int wg_id = blockIdx.x;(wg_id < 256);wg_id = (wg_id + gridDim.x)) {
  int wg_id = blockIdx.x;
  /* mapLocal */
  for (int l_id = threadIdx.x; (l_id < 32); l_id = (l_id + blockDim.x))
  {
      /* oclReduceSeq */
      {
        float x;
        x = 0.0f;
        /* unrolling loop of 2 */
        x = (x + d_input[((2 * l_id) + (64 * wg_id))]);
        x = (x + d_input[((1 + (2 * l_id)) + (64 * wg_id))]);
        up1[l_id] = x;
      }
     
    }

    __syncthreads();
    /* mapLocal */
    for (int l_id = threadIdx.x;(l_id < 16);l_id = (l_id + blockDim.x)) {
      /* oclReduceSeq */
      {
        float x;
        x = 0.0f;
        /* unrolling loop of 2 */
        x = (x + up1[(2 * l_id)]);
        x = (x + up1[(1 + (2 * l_id))]);
        up2[l_id] = x;
      }
     
    }
   
    __syncthreads();
    /* mapLocal */
    for (int l_id = threadIdx.x;(l_id < 8);l_id = (l_id + blockDim.x)) {
      /* oclReduceSeq */
      {
        float x;
        x = 0.0f;
        /* unrolling loop of 2 */
        x = (x + up2[(2 * l_id)]);
        x = (x + up2[(1 + (2 * l_id))]);
        up3[l_id] = x;
      }
     
    }
   
    __syncthreads();
    /* mapLocal */
    for (int l_id = threadIdx.x;(l_id < 4);l_id = (l_id + blockDim.x)) {
      /* oclReduceSeq */
      {
        float x;
        x = 0.0f;
        /* unrolling loop of 2 */
        x = (x + up3[(2 * l_id)]);
        x = (x + up3[(1 + (2 * l_id))]);
        up4[l_id] = x;
      }
     
    }
   
    __syncthreads();
    /* mapLocal */
    for (int l_id = threadIdx.x;(l_id < 2);l_id = (l_id + blockDim.x)) {
      /* oclReduceSeq */
      {
        float x;
        x = 0.0f;
        /* unrolling loop of 2 */
        x = (x + up4[(2 * l_id)]);
        x = (x + up4[(1 + (2 * l_id))]);
        up5[l_id] = x;
      }
     
    }
   
    __syncthreads();
    /* mapLocal */
    for (int l_id = threadIdx.x;(l_id < 1);l_id = (l_id + blockDim.x)) {
      /* oclReduceSeq */
      {
        float x;
        x = 0.0f;
        /* unrolling loop of 2 */
        x = (x + up5[(2 * l_id)]);
        x = (x + up5[(1 + (2 * l_id))]);
        up6[l_id] = x;
      }
     
    }
   
    /* mapSeq */
    /* unrolling loop of 1 */
    d_block_sums[blockIdx.x] = up6[0];
    down1[0] = 0.0f;
    /* mapLocal */
    for (int l_id = threadIdx.x;(l_id < 1);l_id = (l_id + blockDim.x)) {
      /* oclScanSeq */
      {
        float x;
        x = down1[l_id];
        /* unrolling loop of 1 */
        {
          int i = ((int)0);
          down2[(i + (2 * l_id))] = x;
        }
       
        {
          int idx = ((int)0);
          x = (up5[(idx + (2 * l_id))] + x);
        }
       
        down2[(1 + (2 * l_id))] = x;
      }
     
    }
   
    __syncthreads();
    /* mapLocal */
    for (int l_id = threadIdx.x;(l_id < 2);l_id = (l_id + blockDim.x)) {
      /* oclScanSeq */
      {
        float x;
        x = down2[l_id];
        /* unrolling loop of 1 */
        {
          int idxAcc1329 = ((int)0);
          down3[(idxAcc1329 + (2 * l_id))] = x;
        }
       
        {
          int idx = ((int)0);
          x = (up4[(idx + (2 * l_id))] + x);
        }
       
        down3[(1 + (2 * l_id))] = x;
      }
     
    }
   
    __syncthreads();
    /* mapLocal */
    for (int l_id = threadIdx.x;(l_id < 4);l_id = (l_id + blockDim.x)) {
      /* oclScanSeq */
      {
        float x;
        x = down3[l_id];
        /* unrolling loop of 1 */
        {
          int idxAcc1332 = ((int)0);
          down4[(idxAcc1332 + (2 * l_id))] = x;
        }
       
        {
          int idx1333 = ((int)0);
          x = (up3[(idx1333 + (2 * l_id))] + x);
        }
       
        down4[(1 + (2 * l_id))] = x;
      }
     
    }
   
    __syncthreads();
    /* mapLocal */
    for (int l_id_1334 = threadIdx.x;(l_id_1334 < 8);l_id_1334 = (l_id_1334 + blockDim.x)) {
      /* oclScanSeq */
      {
        float x1105;
        x1105 = down4[l_id_1334];
        /* unrolling loop of 1 */
        {
          int idxAcc1335 = ((int)0);
          down5[(idxAcc1335 + (2 * l_id_1334))] = x1105;
        }
       
        {
          int idx1336 = ((int)0);
          x1105 = (up2[(idx1336 + (2 * l_id_1334))] + x1105);
        }
       
        down5[(1 + (2 * l_id_1334))] = x1105;
      }
     
    }
   
    __syncthreads();
    /* mapLocal */
    for (int l_id_1337 = threadIdx.x;(l_id_1337 < 16);l_id_1337 = (l_id_1337 + blockDim.x)) {
      /* oclScanSeq */
      {
        float x1083;
        x1083 = down5[l_id_1337];
        /* unrolling loop of 1 */
        {
          int idxAcc1338 = ((int)0);
          down6[(idxAcc1338 + (2 * l_id_1337))] = x1083;
        }
       
        {
          int idx1339 = ((int)0);
          x1083 = (up1[(idx1339 + (2 * l_id_1337))] + x1083);
        }
       
        down6[(1 + (2 * l_id_1337))] = x1083;
      }
     
    }
   
    __syncthreads();
    /* mapLocal */
    for (int l_id_1340 = threadIdx.x;(l_id_1340 < 32);l_id_1340 = (l_id_1340 + blockDim.x)) {
      /* oclScanSeq */
      {
        float x1061;
        x1061 = down6[l_id_1340];
        /* unrolling loop of 1 */
        {
          int idxAcc1341 = ((int)0);
          output[((idxAcc1341 + (2 * l_id_1340)) + (64 * wg_id))] = x1061;
        }
       
        {
          int idx1342 = ((int)0);
          x1061 = (d_input[((idx1342 + (2 * l_id_1340)) + (64 * wg_id))] + x1061);
        }
       
        output[((1 + (2 * l_id_1340)) + (64 * wg_id))] = x1061;
      }
     
    }
   
    __syncthreads();
  //}
 
}
